
#include <hip/hip_runtime.h>
#define ROTATE_RIGHT(a, rotate) ((a >> rotate) | (a << (sizeof(a) * 8 - rotate)))
typedef unsigned int uint32_t;

extern "C" __global__ void sha256(uint32_t i[64], uint32_t w[8], const uint32_t k[64])
{
    uint32_t h0 = w[0], a = h0,
             h1 = w[1], b = h1,
             h2 = w[2], c = h2,
             h3 = w[3], d = h3,
             h4 = w[4], e = h4,
             h5 = w[5], f = h5,
             h6 = w[6], g = h6,
             h7 = w[7], h = h7;
    for (int j = 16; j < 64; j++)
    {
        uint32_t s0 = i[j - 15], s1 = i[j - 2];
        i[j] = i[j - 16] + i[j - 7] + (ROTATE_RIGHT(s0, 7) ^ ROTATE_RIGHT(s0, 18) ^ (s0 >> 3)) + (ROTATE_RIGHT(s1, 17) ^ ROTATE_RIGHT(s1, 19) ^ (s1 >> 10));
    }

    for (int j = 0; j < 64; j++)
    {
        uint32_t t1 = h + (ROTATE_RIGHT(e, 6) ^ ROTATE_RIGHT(e, 11) ^ ROTATE_RIGHT(e, 25)) + ((e & f) ^ ((~e) & g)) + k[j] + i[j];
        uint32_t t2 = (ROTATE_RIGHT(a, 2) ^ ROTATE_RIGHT(a, 13) ^ ROTATE_RIGHT(a, 22)) + ((a & b) ^ (a & c) ^ (b & c));

        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }
    i[0] = h0 + a;
    i[1] = h1 + b;
    i[2] = h2 + c;
    i[3] = h3 + d;
    i[4] = h4 + e;
    i[5] = h5 + f;
    i[6] = h6 + g;
    i[7] = h7 + h;
}