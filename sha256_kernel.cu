
#include <hip/hip_runtime.h>
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;

__constant__ uint32_t K[64] = {
    0x428a2f98,
    0x71374491,
    0xb5c0fbcf,
    0xe9b5dba5,
    0x3956c25b,
    0x59f111f1,
    0x923f82a4,
    0xab1c5ed5,
    0xd807aa98,
    0x12835b01,
    0x243185be,
    0x550c7dc3,
    0x72be5d74,
    0x80deb1fe,
    0x9bdc06a7,
    0xc19bf174,
    0xe49b69c1,
    0xefbe4786,
    0x0fc19dc6,
    0x240ca1cc,
    0x2de92c6f,
    0x4a7484aa,
    0x5cb0a9dc,
    0x76f988da,
    0x983e5152,
    0xa831c66d,
    0xb00327c8,
    0xbf597fc7,
    0xc6e00bf3,
    0xd5a79147,
    0x06ca6351,
    0x14292967,
    0x27b70a85,
    0x2e1b2138,
    0x4d2c6dfc,
    0x53380d13,
    0x650a7354,
    0x766a0abb,
    0x81c2c92e,
    0x92722c85,
    0xa2bfe8a1,
    0xa81a664b,
    0xc24b8b70,
    0xc76c51a3,
    0xd192e819,
    0xd6990624,
    0xf40e3585,
    0x106aa070,
    0x19a4c116,
    0x1e376c08,
    0x2748774c,
    0x34b0bcb5,
    0x391c0cb3,
    0x4ed8aa4a,
    0x5b9cca4f,
    0x682e6ff3,
    0x748f82ee,
    0x78a5636f,
    0x84c87814,
    0x8cc70208,
    0x90befffa,
    0xa4506ceb,
    0xbef9a3f7,
    0xc67178f2};

#define ROTATE_RIGHT(a, rotate) ((a >> rotate) | (a << (sizeof(a) * 8 - rotate)))
#define COMPRESSION(buffer, output, initial, a, b, c, d, e, f, g, h)                                                                                                            \
    do                                                                                                                                                                          \
    {                                                                                                                                                                           \
        a = initial[0];                                                                                                                                                         \
        b = initial[1];                                                                                                                                                         \
        c = initial[2];                                                                                                                                                         \
        d = initial[3];                                                                                                                                                         \
        e = initial[4];                                                                                                                                                         \
        f = initial[5];                                                                                                                                                         \
        g = initial[6];                                                                                                                                                         \
        h = initial[7];                                                                                                                                                         \
        for (int j = 16; j < 64; j++)                                                                                                                                           \
        {                                                                                                                                                                       \
            uint32_t s0 = buffer[j - 15], s1 = buffer[j - 2];                                                                                                                   \
            buffer[j] = buffer[j - 16] + buffer[j - 7] + (ROTATE_RIGHT(s0, 7) ^ ROTATE_RIGHT(s0, 18) ^ (s0 >> 3)) + (ROTATE_RIGHT(s1, 17) ^ ROTATE_RIGHT(s1, 19) ^ (s1 >> 10)); \
        }                                                                                                                                                                       \
                                                                                                                                                                                \
        for (int j = 0; j < 64; j++)                                                                                                                                            \
        {                                                                                                                                                                       \
            uint32_t t1 = h + (ROTATE_RIGHT(e, 6) ^ ROTATE_RIGHT(e, 11) ^ ROTATE_RIGHT(e, 25)) + ((e & f) ^ ((~e) & g)) + K[j] + buffer[j];                                     \
            uint32_t t2 = (ROTATE_RIGHT(a, 2) ^ ROTATE_RIGHT(a, 13) ^ ROTATE_RIGHT(a, 22)) + ((a & b) ^ (a & c) ^ (b & c));                                                     \
                                                                                                                                                                                \
            h = g;                                                                                                                                                              \
            g = f;                                                                                                                                                              \
            f = e;                                                                                                                                                              \
            e = d + t1;                                                                                                                                                         \
            d = c;                                                                                                                                                              \
            c = b;                                                                                                                                                              \
            b = a;                                                                                                                                                              \
            a = t1 + t2;                                                                                                                                                        \
        }                                                                                                                                                                       \
        output[0] = initial[0] + a;                                                                                                                                             \
        output[1] = initial[1] + b;                                                                                                                                             \
        output[2] = initial[2] + c;                                                                                                                                             \
        output[3] = initial[3] + d;                                                                                                                                             \
        output[4] = initial[4] + e;                                                                                                                                             \
        output[5] = initial[5] + f;                                                                                                                                             \
        output[6] = initial[6] + g;                                                                                                                                             \
        output[7] = initial[7] + h;                                                                                                                                             \
    } while (0)

__device__ void _sha256(uint32_t input[64], uint32_t output[64], const uint32_t initial[8], const uint32_t worker[8])
{
    uint32_t a, b, c, d, e, f, g, h;
    COMPRESSION(input, output, worker, a, b, c, d, e, f, g, h);
    COMPRESSION(output, output, initial, a, b, c, d, e, f, g, h);
}

extern "C" __global__ void sha256(const uint32_t io[16], uint32_t w[8], uint8_t target[32], const uint32_t i[8], uint32_t finished[1])
{
    uint32_t initial = blockIdx.x * blockDim.x + threadIdx.x, nonce = initial;
    uint32_t step = gridDim.x * blockDim.x;

    uint32_t _in[64];
    memcpy(_in, io, 16 * 4);
    uint32_t _out[64];
    uint8_t *_bout = (uint8_t *)_out;
    _out[8] = 1 << 31;
    memset(_out + 9, 0, (15 - 9) * 4);
    _out[15] = 256;
    while (finished[0] == 0)
    {
        _in[3] = nonce;
        _sha256(_in, _out, i, w);
        for (int b = 31; b >= 0; b--)
        {
            if (_bout[b] < target[b] && false)
            {
                if (atomicAdd(finished, 1) == 0)
                {
                    memcpy(w, _out, 7 * 4);
                    w[7] = nonce;
                }
                return;
            }
            else if (_bout[b] > target[b])
                break;
        }
        nonce += step;
        // this means an overflow occurred
        if (nonce < step)
            return;
    }
}